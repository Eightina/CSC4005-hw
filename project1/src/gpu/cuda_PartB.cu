#include "hip/hip_runtime.h"
//
// Created by Zhong Yebin on 2023/9/16.
// Email: yebinzhong@link.cuhk.edu.cn
//
// CUDA implementation of transforming a JPEG image from RGB to gray
//

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h> // CUDA Header

#include "utils.hpp"

// CUDA kernel functon：RGB to Gray

__device__ void rbgarray_filtering (
    unsigned char r_array[],
    unsigned char g_array[],
    unsigned char b_array[],
    JPEGMeta& input_jpeg,
    const unsigned char* input_buffer,
    int loc,
    float* filter,
    int filter_offset
) {
    for (int width = 1; width < input_jpeg.width - 1; ++width) {
        r_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset]);
        g_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset]);
        b_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset]);
        r_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset + 1]);
        g_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset + 1]);
        b_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset + 1]);          
        r_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset + 2]);
        g_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset + 2]);
        b_array[width] += (unsigned char)(input_buffer[loc++] * filter[filter_offset + 2]);
        loc -= 2 * input_jpeg.num_channels;            
    }
}

__global__ void rgbRoutine(
    JPEGMeta& input_jpeg,
    const unsigned char* input_buffer,
    unsigned char* output_buffer,
    float* filter
) {
    int height = blockIdx.x * blockDim.x + threadIdx.x;
    if (height == 0 || height >= input_jpeg.height) {
        return;
    }
    // int end_row = ((start_row + num_rows) > input_jpeg.height) ? input_jpeg.height : (start_row + num_rows);
    // [start_row, end_row)
    // for (int height = start_row; height < end_row; height++) {
    unsigned char *r_array, *g_array, *b_array;
    hipMalloc(&r_array, input_jpeg.width * sizeof(unsigned char));
    hipMalloc(&g_array, input_jpeg.width * sizeof(unsigned char));
    hipMalloc(&b_array, input_jpeg.width * sizeof(unsigned char));

    int rloc = ((height - 1) * input_jpeg.width) * input_jpeg.num_channels;
    rbgarray_filtering(r_array, g_array, b_array,
                        input_jpeg, input_buffer, rloc, filter, 0);

    rloc = ((height) * input_jpeg.width) * input_jpeg.num_channels;
    rbgarray_filtering(r_array, g_array, b_array,
                        input_jpeg, input_buffer, rloc, filter, 3);

    rloc = ((height + 1) * input_jpeg.width) * input_jpeg.num_channels;
    rbgarray_filtering(r_array, g_array, b_array,
                        input_jpeg, input_buffer, rloc, filter, 6);

    for (int width = 1; width < input_jpeg.width - 1; ++width) {
        const int insert_loc = (height * input_jpeg.width + width) * input_jpeg.num_channels;
        output_buffer[insert_loc] = r_array[width];
        output_buffer[insert_loc + 1] = g_array[width];
        output_buffer[insert_loc + 2] = b_array[width];
        // } 
    }
    return;
}

// __global__ void rgbToGray(const unsigned char* input, unsigned char* output,
//                           int width, int height, int num_channels) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < width * height)
//     {
//         unsigned char r = input[idx * num_channels];
//         unsigned char g = input[idx * num_channels + 1];
//         unsigned char b = input[idx * num_channels + 2];
//         output[idx] = static_cast<unsigned char>(0.299 * r + 0.587 * g + 0.114 * b);
//     }
// }

int main(int argc, char** argv) {
    // Verify input argument format
    float filter[9] = {1.0/9, 1.0/9, 1.0/9, 
                        1.0/9, 1.0/9, 1.0/9, 
                        1.0/9, 1.0/9, 1.0/9};
    float* filter_t = &(filter[0]);

    if (argc != 3)
    {
        std::cerr << "Invalid argument, should be: ./executable "
                     "/path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }

    // Read from input JPEG
    const char* input_filepath = argv[1];
    std::cout << "Input file from: " << input_filepath << "\n";
    auto input_jpeg = read_from_jpeg(input_filepath);

    // Allocate memory on host (CPU)
    auto filteredImage = new unsigned char[input_jpeg.width * input_jpeg.height * input_jpeg.num_channels];
    for (int i = 0; i < input_jpeg.width * input_jpeg.height * input_jpeg.num_channels; ++i)
        filteredImage[i] = 0;
    
    // Allocate memory on device (GPU)
    unsigned char* d_input;
    unsigned char* d_output;
    hipMalloc((void**)&d_input, input_jpeg.width * input_jpeg.height *
                                     input_jpeg.num_channels *
                                     sizeof(unsigned char));
    hipMalloc((void**)&d_output, input_jpeg.width * input_jpeg.height *
                                     input_jpeg.num_channels *
                                     sizeof(unsigned char));

    // Copy input data from host to device
    hipMemcpy(d_input, input_jpeg.buffer,
               input_jpeg.width * input_jpeg.height *
                input_jpeg.num_channels * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    // Computation: RGB to Gray
    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // int blockSize = 512; 
    // int blockSize = 256; 
    int blockSize = 128; 
    int numBlocks = input_jpeg.height / blockSize + 1; // each thread a line
    // int rowsPerThread = input_jpeg.height / numBlocks / blockSize;

    hipEventRecord(start, 0); // GPU start time
    rgbRoutine<<<numBlocks, blockSize>>>(
        input_jpeg, d_input, d_output, filter_t
    );
    hipEventRecord(stop, 0); // GPU end time
    hipEventSynchronize(stop);
    
    // Print the result of the GPU computation
    hipEventElapsedTime(&gpuDuration, start, stop);
    // Copy output data from device to host
    hipMemcpy(filteredImage, d_output,
               input_jpeg.width * input_jpeg.height *
                input_jpeg.num_channels * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    // Save output JPEG image
    const char* output_filepath = argv[2];
    std::cout << "Output file to: " << output_filepath << "\n";
    JPEGMeta output_jpeg{filteredImage, input_jpeg.width, input_jpeg.height, input_jpeg.num_channels, input_jpeg.color_space};
    if (write_to_jpeg(output_jpeg, output_filepath)) {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }

    // Release allocated memory on device and host
    hipFree(d_input);
    hipFree(d_output);
    delete[] input_jpeg.buffer;
    delete[] filteredImage;
    std::cout << "Transformation Complete!" << std::endl;
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}